#include "hip/hip_runtime.h"
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/videoio.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
            return -1; \
        } \
    } while (0)

using namespace std;
using namespace cv;

// Kernel for converting BGR to Grayscale
__global__ void bgrToGrayscaleKernel(const unsigned char *d_bgr, unsigned char *d_gray,
                                     int width, int height, int pitch_bgr,
                                     int pitch_gray)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Calculate the BGR pixel position
        int bgrIdx = y * pitch_bgr + 3 * x;
        unsigned char b = d_bgr[bgrIdx];
        unsigned char g = d_bgr[bgrIdx + 1];
        unsigned char r = d_bgr[bgrIdx + 2];

        // Calculate the grayscale value
        unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);

        // Write to the grayscale image
        d_gray[y * pitch_gray + x] = gray;
    }
}


// Kernel for Sobel operator to calculate gradients
__global__ void sobelKernel(const unsigned char *d_input, unsigned char *d_output, int width, int height, int pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
        // Sobel kernels for edge detection in the x and y direction
        int gx = -1 * d_input[(y - 1) * pitch + (x - 1)] + 1 * d_input[(y - 1) * pitch + (x + 1)] +
                 -2 * d_input[y * pitch + (x - 1)] + 2 * d_input[y * pitch + (x + 1)] +
                 -1 * d_input[(y + 1) * pitch + (x - 1)] + 1 * d_input[(y + 1) * pitch + (x + 1)];

        int gy = -1 * d_input[(y - 1) * pitch + (x - 1)] + -2 * d_input[(y - 1) * pitch + x] +
                 -1 * d_input[(y - 1) * pitch + (x + 1)] +
                 1 * d_input[(y + 1) * pitch + (x - 1)] + 2 * d_input[(y + 1) * pitch + x] +
                 1 * d_input[(y + 1) * pitch + (x + 1)];

        int magnitude = sqrtf(gx * gx + gy * gy);
        magnitude = min(max(magnitude, 0), 255); // Clamp to [0, 255]
        d_output[y * pitch + x] = static_cast<unsigned char>(magnitude);
    }
}

// Kernel for non-maximum suppression (NMS)
__global__ void
nonMaxSuppression(const unsigned char *d_input, unsigned char *d_output, int width, int height, int pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
        // Compare the pixel value to its neighbors (8-connectivity)
        if (d_input[y * pitch + x] > d_input[(y - 1) * pitch + x] &&
            d_input[y * pitch + x] > d_input[(y + 1) * pitch + x] &&
            d_input[y * pitch + x] > d_input[y * pitch + (x - 1)] &&
            d_input[y * pitch + x] > d_input[y * pitch + (x + 1)])
        {
            d_output[y * pitch + x] = d_input[y * pitch + x];  // Retain the pixel value
        }
        else
        {
            d_output[y * pitch + x] = 0;  // Set to 0 if it’s not a local maximum
        }
    }
}

// Kernel for applying thresholding
__global__ void applyThreshold(const unsigned char *d_input, unsigned char *d_output, int width, int height, int pitch,
                               int lowThreshold, int highThreshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
        int value = d_input[y * pitch + x];
        if (value >= highThreshold)
        {
            d_output[y * pitch + x] = 255;  // Strong edge
        }
        else if (value >= lowThreshold)
        {
            d_output[y * pitch + x] = 125;  // Weak edge
        }
        else
        {
            d_output[y * pitch + x] = 0;  // Non-edge
        }
    }
}

// Add logging for time measurement
void logTime(const string &message, const chrono::steady_clock::time_point &start)
{
    auto end = chrono::steady_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();
    cout << message << " took " << duration << " ms" << endl;
}



// Main function to process video using CUDA
int main(int argc, char *argv[])
{
    if (argc < 2)
    {
        cerr << "Usage: " << argv[0] << " <video_path>" << endl;

        return -1;
    }

    string videoPath = argv[1];
    cv::VideoCapture cap(videoPath);

    if (!cap.isOpened())
    {
        cerr << "Error: Couldn't open video file: " << videoPath << endl;
        return -1;
    }

    cout << "Processing video: " << videoPath << endl;

    const string window_name = "Sobel Edge Detection with CUDA";
    cv::namedWindow(window_name);

    int lowThreshold = 50;   // Low threshold for Canny
    int highThreshold = 150; // High threshold for Canny
    int frameWidth = static_cast<int>(cap.get(cv::CAP_PROP_FRAME_WIDTH));
    int frameHeight = static_cast<int>(cap.get(cv::CAP_PROP_FRAME_HEIGHT));

    size_t pitch_bgr, pitch_gray;
    unsigned char *d_bgr = nullptr, *d_gray = nullptr, *d_output = nullptr, *d_result = nullptr;
    unsigned char *h_result = new unsigned char[frameWidth * frameHeight];

    // Logging memory allocation
    auto start = chrono::steady_clock::now();
    cout << "Allocating GPU memory..." << endl;

    CUDA_CHECK(hipMallocPitch(&d_bgr, &pitch_bgr, 3 * frameWidth * sizeof(unsigned char), frameHeight));
    CUDA_CHECK(hipMallocPitch(&d_gray, &pitch_gray, frameWidth * sizeof(unsigned char), frameHeight));
    CUDA_CHECK(hipMallocPitch(&d_output, &pitch_gray, frameWidth * sizeof(unsigned char), frameHeight));
    CUDA_CHECK(hipMallocPitch(&d_result, &pitch_gray, frameWidth * sizeof(unsigned char), frameHeight));

    logTime("Memory allocation", start);

    // Define CUDA block and grid sizes
    dim3 block(16, 16);
    dim3 grid((frameWidth + block.x - 1) / block.x, (frameHeight + block.y - 1) / block.y);


    cv::Mat frame;
    while (true)
    {
        start = chrono::steady_clock::now();

        cap >> frame;
        if (frame.empty()) break;

        cout << "Processing frame..." << endl;

        // Copy the BGR frame to GPU memory
        CUDA_CHECK(hipMemcpy2D(d_bgr, pitch_bgr, frame.data, frame.step, 3 * frameWidth, frameHeight,
                                hipMemcpyHostToDevice));
        logTime("Copying frame to GPU", start);

        start = chrono::steady_clock::now();
        // Step 1: Convert BGR to grayscale using CUDA kernel
        bgrToGrayscaleKernel<<<grid, block>>>(d_bgr, d_gray, frameWidth, frameHeight, pitch_bgr, pitch_gray);
        CUDA_CHECK(hipPeekAtLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        logTime("Grayscale conversion", start);

        start = chrono::steady_clock::now();
        // Step 2: Sobel edge detection kernel (gradient calculation)
        sobelKernel<<<grid, block>>>(d_gray, d_output, frameWidth, frameHeight, pitch_gray);
        CUDA_CHECK(hipPeekAtLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Step 3: Non-maximum suppression kernel
        nonMaxSuppression<<<grid, block>>>(d_output, d_result, frameWidth, frameHeight, pitch_gray);
        CUDA_CHECK(hipPeekAtLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Step 4: Apply thresholds to get final edge detection result
        applyThreshold<<<grid, block>>>(d_result, d_output, frameWidth, frameHeight, pitch_gray, lowThreshold,
                                        highThreshold);
        CUDA_CHECK(hipPeekAtLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        logTime("Sobel kernel", start);

        start = chrono::steady_clock::now();
        // Copy the result back to the host
        CUDA_CHECK(hipMemcpy2D(h_result, frameWidth, d_output, pitch_gray, frameWidth, frameHeight,
                                hipMemcpyDeviceToHost));
        logTime("Copying result to CPU", start);

        // Create a Mat object to display the result
        cv::Mat outputFrame(frameHeight, frameWidth, CV_8UC1, h_result);
        cv::imshow(window_name, outputFrame);

        auto key = (char) waitKey(30);
        if (key == 'q' || key == 27)
        {
            cout << "Processing stopped by user. Exiting..." << endl;
            break;
        }
    }


    // Cleanup
    CUDA_CHECK(hipFree(d_bgr));
    CUDA_CHECK(hipFree(d_gray));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_result));
    delete[] h_result;

    cout << "Cleanup completed. Goodbye!" << endl;

    return 0;
}